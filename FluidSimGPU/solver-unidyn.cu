#include "hip/hip_runtime.h"
#include "visit_writer.h"
#include <math.h>
#include <cmath>
#include <sstream>
#include <iostream>
#include <cstdlib>
#include <random>
#include <chrono>
#include <algorithm>
#include <future>
#include <vector>
#include "FluidGPU-unidyn.cuh"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>




const int nspts = 20100; //number of solid particles
const int nbpts = 400;//1000; //number of solid particles
const int tpts = 101;

//Storage for output
int vardims3[] = { 1,1 };
int morton_host(unsigned int x, unsigned int y, unsigned int z) {
	//int x = (bidx / GRIDSIZE / GRIDSIZE);
	//int y = (bidx / GRIDSIZE % GRIDSIZE);
	//int z = (bidx % GRIDSIZE);

	x = (x | (x << 16)) & 0x030000FF;
	x = (x | (x << 8)) & 0x0300F00F;
	x = (x | (x << 4)) & 0x030C30C3;
	x = (x | (x << 2)) & 0x09249249;

	y = (y | (y << 16)) & 0x030000FF;
	y = (y | (y << 8)) & 0x0300F00F;
	y = (y | (y << 4)) & 0x030C30C3;
	y = (y | (y << 2)) & 0x09249249;

	z = (z | (z << 16)) & 0x030000FF;
	z = (z | (z << 8)) & 0x0300F00F;
	z = (z | (z << 4)) & 0x030C30C3;
	z = (z | (z << 2)) & 0x09249249;

	return x | (y << 1) | (z << 2);

}

int demorton_host(unsigned int x, int b) {
	//b should be 0 for x, 1 for y, 2 for z
	switch (b) {
	case 0: break;
	case 1: x = (x >> 1);
		break;
	case 2: x = (x >> 2);
		break;
	}
	x &= 0x09249249;                  // x = ---- 9--8 --7- -6-- 5--4 --3- -2-- 1--0
	x = (x | (x >> 2)) & 0x030c30c3; // x = ---- --98 ---- 76-- --54 ---- 32-- --10
	x = (x | (x >> 4)) & 0x0300f00f; // x = ---- --98 ---- ---- 7654 ---- ---- 3210
	x = (x | (x >> 8)) & 0xff0000ff; // x = ---- --98 ---- ---- ---- ---- 7654 3210
	x = (x | (x >> 16)) & 0x000003ff; // x = ---- ---- ---- ---- ---- --98 7654 3210
	return x;
}

int main(int argc, char **argv)
{
	/*
	std::cout << morton(30, 30, 30) << "\n";
	for (int k = -1; k < 2; k++)
		for (int j = -1; j < 2; j++)
			for (int i = -1; i < 2; i++)
				std::cout << morton(demorton(morton(30 + i, 30 + j, 30 + k), 0), demorton(morton(30 + i, 30 + j, 30 + k), 1), demorton(morton(30 + i, 30 + j, 30 + k), 2)) << "\n";
				*/
    //hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    int dev;

	size_t size = (nspts+nbpts) * sizeof(Particle);
	//Particle *SPptr[deviceCount];
	Particle *d_SPptr[deviceCount];
	std::vector<Particle> SPptr[2];
	Particle *d_Pbuff[deviceCount];
	int *d_cbuff[deviceCount];
    
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        //SPptr[dev] = (Particle *)malloc(size); // Allocate particles on host
        SPptr[dev].reserve(nspts+nbpts);
		hipMalloc((void **)&d_SPptr[dev], size); // Allocate particles on device
		hipMalloc((void **)&d_Pbuff[dev], (nspts+nbpts)*sizeof(Particle)); // Allocate particles on device
    }

   
    float *spts[deviceCount];
    float *a3[deviceCount];
    float *b3[deviceCount];
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipMallocManaged(&spts[dev], 3*(nspts + nbpts) * sizeof(float));
        hipMallocManaged(&a3[dev], (nspts + nbpts) * sizeof(float));
        hipMallocManaged(&b3[dev], (nspts + nbpts) * sizeof(float));
    

	    for (int i = 0; i < 3*(nspts + nbpts); i++){
            spts[dev][i] =0 ;
        }
    }



	const char * const varnames3[] = { "subindex", "cellnumber" };
	float *arraysGPU1[] = { (float*)a3[0], (float*)b3[0], };  //only use one GPU for writing for now
    float *arraysGPU2[] = { (float*)a3[1], (float*)b3[1], };  //only use one GPU for writing for now
  

	//Set up Solid Particles
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
	    for (int j = 0; j < nspts; j++) {
		    	SPptr[dev].push_back(Particle(-.6 + 0.04*((j / 20) % 20), -0.6 + 0.04*(j % 20), -0.80 +(j / 20 /20)*0.04, 0., 0., 0.));
		    	SPptr[dev][j].index = j;
		    	SPptr[dev][j].solid = 0;
		    	SPptr[dev][j].fluid = 1;
		    	SPptr[dev][j].dens = RHO_0;
		    	SPptr[dev][j].cellnumber = int((SPptr[dev][j].xcoord - XMIN) / CELLSIZE)*GRIDSIZE*GRIDSIZE + int((SPptr[dev][j].ycoord - YMIN) / CELLSIZE)*GRIDSIZE + int((SPptr[dev][j].zcoord - ZMIN) /CELLSIZE);
		    	//SPptr[j].cellnumber = morton_host(int((SPptr[j].xcoord - XMIN) / CELLSIZE), int((SPptr[j].ycoord - YMIN) / CELLSIZE), int((SPptr[j].zcoord - ZMIN)/CELLSIZE));

			}
/*
	    for (int j = 29200; j < nspts; j++) {
	    	SPptr[dev].push_back(Particle(-.5+0.05*(((j- 29200) / 10) % 10), -.5 + 0.05*((j - 29200) % 10), 0.75 + ((j - 29200) / 100)*0.05, 0., 0., 0.));
	    	SPptr[dev][j].index = j;
	    	SPptr[dev][j].solid = 0;
	    	SPptr[dev][j].fluid = 1;
	    	SPptr[dev][j].dens = RHO_0;
	    	SPptr[dev][j].cellnumber = int((SPptr[dev][j].xcoord - XMIN) / CELLSIZE)*GRIDSIZE*GRIDSIZE + int((SPptr[dev][j].ycoord - YMIN) / CELLSIZE)*GRIDSIZE + int((SPptr[dev][j].zcoord - ZMIN) / CELLSIZE);
		    //SPptr[j].cellnumber = morton_host(int((SPptr[j].xcoord - XMIN) / CELLSIZE), int((SPptr[j].ycoord - YMIN) / CELLSIZE), int((SPptr[j].zcoord - ZMIN)/CELLSIZE));
	    }
*/
	    //Set up boundary particles
	    for (int i = 0; i < nbpts; i++) {
		    SPptr[dev].push_back(Particle(-0.6 + 0.06*(i % 20), -0.6 + 0.06*(i / 20), -0.96, true));
	    	SPptr[dev][nspts + i].index = nspts+i;
	    	SPptr[dev][nspts + i].solid = 1;
	    	SPptr[dev][nspts + i].fluid = 0;
	    	SPptr[dev][nspts + i].dens = RHO_0_SAND;
	    	SPptr[dev][nspts + i].cellnumber = int((SPptr[dev][i + nspts].xcoord - XMIN) / CELLSIZE)*GRIDSIZE*GRIDSIZE + int((SPptr[dev][i + nspts].ycoord - YMIN) / CELLSIZE)*GRIDSIZE + int((SPptr[dev][i + nspts].zcoord - ZMIN) / CELLSIZE);
			//SPptr[i+nspts].cellnumber = morton_host(int((SPptr[i + nspts].xcoord - XMIN) / CELLSIZE), int((SPptr[i + nspts].ycoord - YMIN) / CELLSIZE), int((SPptr[i + nspts].zcoord - ZMIN) / CELLSIZE));

	    }
/*
	    for (int i = 0; i < nbpts / 8; i++) {
	    	SPptr[dev].push_back(Particle(-0.96 + 0.06*(i % 30), -0.96, -0.94 + 0.06*(i / 30), true));
	    	SPptr[dev][nspts + i + nbpts / 2].index = nspts + i + nbpts / 2;
	    	SPptr[dev][nspts + i + nbpts / 2].solid = 1;
	    	SPptr[dev][nspts + i + nbpts / 2].fluid = 0;
	    	SPptr[dev][nspts + i + nbpts / 2].dens = RHO_0_SAND;
	    	SPptr[dev][nspts + i + nbpts / 2].cellnumber = int((SPptr[dev][nspts + i + nbpts / 2].xcoord - XMIN) / CELLSIZE)*GRIDSIZE*GRIDSIZE + int((SPptr[dev][nspts + i + nbpts / 2].ycoord - YMIN) / CELLSIZE)*GRIDSIZE + int((SPptr[dev][nspts + i + nbpts / 2].zcoord - ZMIN) / CELLSIZE);
		}
		for (int i = 0; i < nbpts / 8; i++) {
	    	SPptr[dev].push_back(Particle(-0.96 + 0.06*(i % 30), 0.84, -0.94 + 0.06*(i / 30), true));
	    	SPptr[dev][nspts + i + 5*nbpts / 8].index = nspts + i + 5 * nbpts / 8;
	    	SPptr[dev][nspts + i + 5*nbpts / 8].solid = 1;
	    	SPptr[dev][nspts + i + 5*nbpts / 8].fluid = 0;
	    	SPptr[dev][nspts + i + 5*nbpts / 8].dens = RHO_0_SAND;
	    	SPptr[dev][nspts + i + 5*nbpts / 8].cellnumber = int((SPptr[dev][nspts + i + 5 * nbpts / 8].xcoord - XMIN) / CELLSIZE)*GRIDSIZE*GRIDSIZE + int((SPptr[dev][nspts + i + 5 * nbpts / 8].ycoord - YMIN) / CELLSIZE)*GRIDSIZE + int((SPptr[dev][nspts + i + 5 * nbpts / 8].zcoord - ZMIN) / CELLSIZE);
		}
		for (int i = 0; i < nbpts / 8; i++) {
	    	SPptr[dev].push_back(Particle(-0.96, -0.96 + 0.06*(i % 30), -0.94 + 0.06*(i / 30), true));
	    	SPptr[dev][nspts + i + 6*nbpts / 8].index = nspts + i + 6 * nbpts / 8;
	    	SPptr[dev][nspts + i + 6*nbpts / 8].solid = 1;
	    	SPptr[dev][nspts + i + 6*nbpts / 8].fluid = 0;
	    	SPptr[dev][nspts + i + 6*nbpts / 8].dens = RHO_0_SAND;
	    	SPptr[dev][nspts + i + 6*nbpts / 8].cellnumber = int((SPptr[dev][nspts + i + 6 * nbpts / 8].xcoord - XMIN) / CELLSIZE)*GRIDSIZE*GRIDSIZE + int((SPptr[dev][nspts + i + 6 * nbpts / 8].ycoord - YMIN) / CELLSIZE)*GRIDSIZE + int((SPptr[dev][nspts + i + 6 * nbpts / 8].zcoord - ZMIN) / CELLSIZE);
		}
		for (int i = 0; i < nbpts / 8; i++) {
    		SPptr[dev].push_back(Particle(0.78, -0.96 + 0.06*(i % 30), -0.94 + 0.06*(i / 30), true));
    		SPptr[dev][nspts + i + 7*nbpts / 8].index = nspts + i + 7 * nbpts / 8;
    		SPptr[dev][nspts + i + 7*nbpts / 8].solid = 1;
    		SPptr[dev][nspts + i + 7*nbpts / 8].fluid = 0;
    		SPptr[dev][nspts + i + 7*nbpts / 8].dens = RHO_0_SAND;
    		SPptr[dev][nspts + i + 7*nbpts / 8].cellnumber = int((SPptr[dev][nspts + i + 7 * nbpts / 8].xcoord - XMIN) / CELLSIZE)*GRIDSIZE*GRIDSIZE + int((SPptr[dev][nspts + i + 7 * nbpts / 8].ycoord - YMIN) / CELLSIZE)*GRIDSIZE + int((SPptr[dev][nspts + i + 7 * nbpts / 8].zcoord - ZMIN) / CELLSIZE);
		
	    	//SPptr[i+nspts].cellnumber = morton_host(int((SPptr[i + nspts].xcoord - XMIN) / CELLSIZE), int((SPptr[i + nspts].ycoord - YMIN) / CELLSIZE), int((SPptr[i + nspts].zcoord - ZMIN) / CELLSIZE));
        }*/
	}
	int buffer = GRIDSIZE*GRIDSIZE; 
	//const int Ns[2] = {(NUMCELLS)/2, NUMCELLS - (NUMCELLS)/2 };
	
	//test single gpu
	const int Ns[2] = {(NUMCELLS), 0 };
	deviceCount = 1;
	buffer = 0;
	
	
	for (dev = 0; dev< deviceCount;dev++){
		if (deviceCount>1){
        for (int j = 0; j < SPptr[dev].size(); j++) {
            if (SPptr[dev][j].cellnumber < Ns[0] -buffer && dev == 1){
                SPptr[dev].erase(SPptr[dev].begin() + j);
                j--;
            }
            if (SPptr[dev][j].cellnumber >= Ns[0] +buffer && dev == 0){
                SPptr[dev].erase(SPptr[dev].begin() + j);
                j--;
            }
            //std::cout << "Cellnumber = " << SPptr[dev][j].cellnumber << ", dev = " << dev << "\n";
		}
	}
        hipMemcpy(d_SPptr[dev], &SPptr[dev][0], SPptr[dev].size() * sizeof(Particle), hipMemcpyHostToDevice);
	}
	

	///////Sort particles by cell number and keep track of when a new cell starts//////////
	std::vector<int> v_h[deviceCount];
	std::vector<int> particleindex[deviceCount]; 
	int *v_d[deviceCount];
	int *d_particleindex[deviceCount];
	const int N = nspts + nbpts;  // Number of elements in arrays
	size_t sizes = N * sizeof(int);
    
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
		v_h[dev].reserve(N);        // Allocate array on host
		particleindex[dev].reserve(N);        // Allocate array on host
		hipMalloc((void **)&v_d[dev], int(SPptr[dev].size()) * sizeof(int));// Allocate array on device
		hipMalloc((void **)&d_particleindex[dev], int(SPptr[dev].size()) * sizeof(int));// Allocate array on device
	    for (int i = 0; i<SPptr[dev].size(); i++)
	    {
			v_h[dev].push_back(SPptr[dev][i].cellnumber);
			particleindex[dev].push_back(i);
	    }

	hipMemcpy(v_d[dev], &v_h[dev][0], int(SPptr[dev].size()) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_particleindex[dev], &particleindex[dev][0], int(SPptr[dev].size()) * sizeof(int), hipMemcpyHostToDevice);
    //std::cout << hipGetErrorName(hipGetLastError())<< "\n";
	}
	
	int *xleft[deviceCount];// = {0};
    int *xright[deviceCount];//= {0};
    int *sizeleft[deviceCount];//= {0};
    int *sizeright[deviceCount];//= {0};
    int *d_xl[deviceCount];
    int *d_xr[deviceCount];
    int *d_sl[deviceCount];
    int *d_sr[deviceCount];
    for(int dev=0; dev<deviceCount; dev++) {
        hipSetDevice(dev);
        xleft[dev] = (int *)malloc(sizeof(int));
        xright[dev] = (int *)malloc(sizeof(int));
        sizeleft[dev] = (int *)malloc(sizeof(int));
        sizeright[dev] = (int *)malloc(sizeof(int));
        hipMalloc((void **)&d_xl[dev], sizeof(int));// Allocate array on device
        hipMalloc((void **)&d_xr[dev], sizeof(int));// Allocate array on device
        hipMalloc((void **)&d_sl[dev], sizeof(int));// Allocate array on device
        hipMalloc((void **)&d_sr[dev], sizeof(int));// Allocate array on device
    }
/*
    thrust::device_ptr<Particle> t_a[2];
    thrust::device_ptr<int> t_v[2];
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        thrust::device_ptr<Particle> t_b(d_SPptr[dev]);
        thrust::device_ptr<int> t_x(v_d[dev]);
        t_a[dev] = t_b;
        t_v[dev] = t_x;

    }
*/
    
	int *start, *end, *split, *numsplit, *d_start[deviceCount], *d_end[deviceCount],*d_split[deviceCount], *d_numsplit, *start_copy, *d_start_copy[deviceCount];
    size_t sizes2[2] = {(Ns[0] +buffer)* sizeof(int),(Ns[1]+buffer)*sizeof(int)};

	start = (int *)malloc(NUMCELLS*sizeof(int));        // Allocate array on host
	start_copy = (int *)malloc(NUMCELLS*sizeof(int));        // Allocate array on host
	end = (int *)malloc(NUMCELLS*sizeof(int));        // Allocate array on host
	split = (int *)malloc(NUMCELLS*sizeof(int));        // Allocate array on host
	numsplit = (int *)malloc(sizeof(int));        // Allocate array on host
    numsplit[0] = 0;

    for (int i = 0; i<NUMCELLS; i++)
    {
		start[i] = -1;
		start_copy[i] = -1;
		end[i] = -1;
		split[i] = -1;
    }

    
   
    for(int dev=0, pos=0; dev<deviceCount; pos+=Ns[dev], dev++) {
        hipSetDevice(dev);
        hipMalloc((void **)&d_start[dev], sizes2[dev]);// Allocate array on device
		//std::cout << hipGetErrorName(hipGetLastError())<< "\n";
		hipMalloc((void **)&d_start_copy[dev], sizes2[dev]);// Allocate array on device
        //std::cout << hipGetErrorName(hipGetLastError())<< "\n";
        hipMalloc((void **)&d_end[dev], sizes2[dev]);// Allocate array on device
		//std::cout << hipGetErrorName(hipGetLastError())<< "\n";
		hipMalloc((void **)&d_split[dev], sizes2[dev]);// Allocate array on device
		//std::cout << hipGetErrorName(hipGetLastError())<< "\n";
		hipMalloc((void **)&d_numsplit, sizeof(int));// Allocate array on device
        //std::cout << hipGetErrorName(hipGetLastError())<< "\n";

        hipMemcpy(d_start[dev], start+pos-(dev>0)*buffer, sizes2[dev], hipMemcpyHostToDevice);
		//std::cout << hipGetErrorName(hipGetLastError())<< "\n";
		hipMemcpy(d_start_copy[dev], start_copy+pos-(dev>0)*buffer, sizes2[dev], hipMemcpyHostToDevice);
        //std::cout << hipGetErrorName(hipGetLastError())<< "\n";
        hipMemcpy(d_end[dev], end+pos-(dev>0)*buffer, sizes2[dev], hipMemcpyHostToDevice);
		//std::cout << hipGetErrorName(hipGetLastError())<< "\n";
		hipMemcpy(d_split[dev], split+pos-(dev>0)*buffer, sizes2[dev], hipMemcpyHostToDevice);
		//std::cout << hipGetErrorName(hipGetLastError())<< "\n";
		hipMemcpy(d_numsplit, numsplit, sizeof(int), hipMemcpyHostToDevice);
        //std::cout << hipGetErrorName(hipGetLastError())<< "\n";
    }

	int dsz[2] ={int(SPptr[0].size()),int(SPptr[1].size())}; 
	for (int t = 0; t < tpts; t++) {
		std::cout << "t= " << t << "\n";
		//std::cout << SPptr[0].size() << " particles on device 0 at t="<< t<< "\n";
		//std::cout << SPptr[1].size() << " particles on device 1 at t="<< t<< "\n";
		
		hipEvent_t start, stop;
		CUDA_CHECK_RETURN(hipEventCreate(&start));
		CUDA_CHECK_RETURN(hipEventCreate(&stop));
		float elapsedTime;
		CUDA_CHECK_RETURN(hipEventRecord(start, 0));
		hipDeviceSynchronize();
        for (dev = 0; dev < deviceCount; dev++){
			hipSetDevice(dev);
			
			//thrust::device_ptr<Particle> t_b(d_SPptr[dev]);
			thrust::device_ptr<int> t_b(d_particleindex[dev]);
			thrust::device_ptr<int> t_x(v_d[dev]);

			thrust::sort_by_key(t_x, t_x + dsz[dev], t_b);
		   
			if (strcmp(hipGetErrorName(hipGetLastError()),"hipSuccess")!=0){
			//	std::cout << "Sorting failed at t = "<< t << ", " << hipGetErrorName(hipGetLastError())<< "\n";
			}
		}


		
		for (dev = 0; dev < deviceCount; dev++){
			hipSetDevice(dev);
            findneighbours << <NUMCELLS, 1024 >> > (v_d[dev], d_start[dev], d_start_copy[dev], d_end[dev],dsz[dev], dev*(Ns[0]-buffer)); //(sorted list of particles, start cells, end cells, number of particles, start index)
			if (strcmp(hipGetErrorName(hipGetLastError()),"hipSuccess")!=0){
			//	std::cout << "Binning failed at t = "<< t << ", " << hipGetErrorName(hipGetLastError())<< "\n";
			}
		}

		
		for (dev = 0; dev < deviceCount; dev++){
			hipSetDevice(dev);
			mykernel <<<NUMCELLS, 1024 >> > (d_SPptr[dev], d_particleindex[dev], v_d[dev], d_start[dev], d_end[dev], d_split[dev], dsz[dev],Ns[dev]+buffer,dev,buffer,d_numsplit);
			//hipDeviceSynchronize();
			if (strcmp(hipGetErrorName(hipGetLastError()),"hipSuccess")!=0){
				std::cout << "Force calc failed at t = "<< t << ",  " << hipGetErrorName(hipGetLastError())<< "\n";
			}
		}
			hipMemcpy(numsplit, d_numsplit, sizeof(int), hipMemcpyDeviceToHost);
			

			//std::cout<<numsplit[0] << "\n";
			//sort start by split
			thrust::device_ptr<int> t_1(d_start_copy[0]);
			thrust::device_ptr<int> t_2(d_split[0]);
			thrust::sort_by_key(t_2, t_2 + NUMCELLS, t_1,  thrust::greater<int>());

			for (dev = 0; dev < deviceCount; dev++){
				hipSetDevice(dev);
				mykernel3 <<<numsplit[0]*8, 240 >> > (d_SPptr[dev], d_particleindex[dev], v_d[dev], d_start[dev], d_end[dev], d_split[dev], dsz[dev],Ns[dev]+buffer,dev,buffer,d_numsplit);
				hipDeviceSynchronize();
				if (strcmp(hipGetErrorName(hipGetLastError()),"hipSuccess")!=0){
					std::cout << "Force calc #2 failed at t = "<< t << ",  " << hipGetErrorName(hipGetLastError())<< "\n";
				}
			}


		
		for (dev = 0; dev < deviceCount; dev++){
			hipSetDevice(dev);
			mykernel2 << <NUMCELLS, 1024 >> > (d_SPptr[dev], d_particleindex[dev], v_d[dev], d_start_copy[dev], d_start[dev], d_end[dev],d_split[dev],d_numsplit,dsz[dev], Ns[dev]+buffer, dev, buffer, spts[dev], a3[dev], b3[dev]);
			if (strcmp(hipGetErrorName(hipGetLastError()),"hipSuccess")!=0){
			//	std::cout << "Updating failed at t = "<< t << ",  " << hipGetErrorName(hipGetLastError())<< "\n";
			}
		}

		if (deviceCount>1){
		for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
			find_idx << <NUMCELLS, 1024 >> > (v_d[dev], dev, dsz[dev], buffer, d_xl[dev], d_xr[dev], d_sl[dev], d_sr[dev]);
			if (strcmp(hipGetErrorName(hipGetLastError()),"hipSuccess")!=0){
				std::cout << "Index search failed at t = "<< t << ",  " << hipGetErrorName(hipGetLastError())<< "\n";
            }
            hipDeviceSynchronize();
            hipMemcpy(&xleft[dev][0], d_xl[dev],sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&xright[dev][0], d_xr[dev],sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&sizeleft[dev][0], d_sl[dev],sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&sizeright[dev][0], d_sr[dev],sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
		}

		for (dev = 0; dev<deviceCount; dev++){
			dsz[dev] = dev==0 ? (xright[dev][0]-xleft[dev][0]+1) + (sizeleft[(dev+1)][0]-xleft[(dev+1)][0]) : (xright[dev-1][0]-sizeright[dev-1][0]+1) +  (xright[dev][0]-xleft[dev][0]+1);  
		}

		//take sleft-xleft particles from dev x at xleft to dev x-1 at xright+1 of devx-1
		//new max index of dev x-1 is xright_{x-1} + (sleft-xleft)_{x} 

	//use particleindex here later
		dev= 1;
		hipSetDevice(1);
		hipMemcpy(&SPptr[dev-1][xright[dev-1][0]+1],d_SPptr[dev]+xleft[dev][0],(sizeleft[dev][0]-xleft[dev][0])*sizeof(Particle), hipMemcpyDeviceToHost);
		hipSetDevice(0);
		hipMemcpy(d_SPptr[dev-1]+xright[dev-1][0]+1,&SPptr[dev-1][xright[dev-1][0]+1],(sizeleft[dev][0]-xleft[dev][0])*sizeof(Particle), hipMemcpyHostToDevice);

		//how many particles will be transferred to dev x = (xright-sright+1)_{x-1}
		//therefore need to shift dev x particles to the left by xleft_{x}-(xright-sright+1)_{x-1}
		//shift particle indices xleft_x --> xright_{x} + (sleft-xleft)_{x+1} 

//use particleindex here later
        hipSetDevice(1);
        mem_shift << <NUMCELLS, 1024 >> > (d_SPptr[dev], d_Pbuff[dev], v_d[dev], d_cbuff[dev], dev,xleft[dev][0]-(xright[dev-1][0]-sizeright[dev-1][0]+1),xleft[dev][0],xright[dev][0]);
        hipDeviceSynchronize();
        if (strcmp(hipGetErrorName(hipGetLastError()),"hipSuccess")!=0){
			std::cout << "Mem shift failed at t = "<< t << ",  " << hipGetErrorName(hipGetLastError())<< "\n";
        }
            

		
		//now that particles shifted, indices shift as well
		int xleftold[2];
		int xrightold[2];
		int sizeleftold[2];
		int sizerightold[2];
		for (dev = 0; dev<deviceCount; dev++){
			xleftold[dev] = xleft[dev][0];
			xrightold[dev] = xright[dev][0];
			sizeleftold[dev] = sizeleft[dev][0];
			sizerightold[dev] = sizeright[dev][0];
		}
		dev = 1;
		xleft[dev][0] -= xleftold[dev]-(xrightold[dev-1]-sizerightold[dev-1]+1);
		xright[dev][0] -= xleftold[dev]-(xrightold[dev-1]-sizerightold[dev-1]+1);
		sizeleft[dev][0] -= xleftold[dev]-(xrightold[dev-1]-sizerightold[dev-1]+1);
		sizeright[dev][0] -= xleftold[dev]-(xrightold[dev-1]-sizerightold[dev-1]+1);
			
//use particleindex here later
		//take xrightold-srightold+1 particles from dev x at srightnew to dev x+1 at 0
        dev = 0;
		hipSetDevice(dev);
		hipMemcpy(&SPptr[dev+1][0],d_SPptr[dev]+sizeright[dev][0],(xrightold[dev]-sizerightold[dev]+1)*sizeof(Particle), hipMemcpyDeviceToHost);
		hipSetDevice(dev+1);
		hipMemcpy(d_SPptr[dev+1],&SPptr[dev+1][0],(xrightold[dev]-sizerightold[dev]+1)*sizeof(Particle), hipMemcpyHostToDevice);
		


		for (dev = 0;dev<deviceCount;dev++){
			dsz[dev] = dev==0 ? (xright[dev][0]-xleft[dev][0]+1) + (sizeleft[(dev+1)][0]-xleft[(dev+1)][0]) : (xright[dev-1][0]-sizeright[dev-1][0]+1) +  (xright[dev][0]-xleft[dev][0]+1);
		}
		
	}

		if (t % 30 == 0) {
			for (dev = 0; dev < deviceCount; dev++){
				hipSetDevice(dev);
				hipMemcpy(&SPptr[dev][0], d_SPptr[dev], dsz[dev] * sizeof(Particle), hipMemcpyDeviceToHost); //copy updated particles back to cpu
				if (strcmp(hipGetErrorName(hipGetLastError()),"hipSuccess")!=0){
					std::cout << "Particle copy failed at t = "<< t << ",  " << hipGetErrorName(hipGetLastError())<< "\n";
				}
			}
			hipDeviceSynchronize();
			//Write each frame to file
			std::ostringstream oss;
			std::ostringstream oss2;
			oss << "anim-uni/anim_s_GPU0_" << t / 1<< ".vtk";
			std::string var = oss.str();
			const char* cstr = var.c_str();
			write_point_mesh(cstr, 0, dsz[0], spts[0], 2, vardims3, varnames3, arraysGPU1);
			oss2 << "anim-uni/anim_s_GPU1_" << t / 1 << ".vtk";
			std::string var2 = oss2.str();
			const char* cstr2 = var2.c_str();
			//write_point_mesh(cstr2, 0, dsz[1], spts[1], 2, vardims3, varnames3, arraysGPU2);

		}
		//Recalculate cells
		for (dev = 0; dev< deviceCount; dev++){ //take xright-sright particles from dev x at sright to dev x+1 at 0
			hipSetDevice(dev);
			//recalculate sizes of arrays
			cell_calc << <NUMCELLS, 1024 >> >(d_SPptr[dev], d_particleindex[dev], v_d[dev], dsz[dev],dev);
			hipDeviceSynchronize();
		}



		hipDeviceSynchronize();
		CUDA_CHECK_RETURN(hipEventRecord(stop, 0));

		CUDA_CHECK_RETURN(hipEventSynchronize(stop));
		CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
		CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
		CUDA_CHECK_RETURN(hipGetLastError());
		CUDA_CHECK_RETURN(hipEventDestroy(start));
		CUDA_CHECK_RETURN(hipEventDestroy(stop));
		std::cout << "\nElapsed kernel time: " << elapsedTime << " ms\n";


		//std::cout << hipGetErrorName(hipGetLastError()) << "\n";
		

//		hipDeviceSynchronize();

		
	
	}
	return 0;
}
